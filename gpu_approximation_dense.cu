#include "hip/hip_runtime.h"
#include <omp.h>
#include <stdio.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
//#include "util.h"
#include "flags.h"
#include "gpu_wrappers.h"
using namespace std;

static int glob_nov;
static int glob_sizeof_c;
static int glob_sizeof_s;

//Utilities run on CPU//
template <class C, class S>
bool ScaleMatrix(S M,
		 int nov,
		 long row_extracted,
		 long col_extracted,
		 C d_r[],
		 C d_c[],
		 int scale_times) {
  
  for (int k = 0; k < scale_times; k++) {
    
    for (int j = 0; j < nov; j++) {
      if (!((col_extracted >> j) & 1L)) {
	C col_sum = 0;
	for (int i = 0; i < nov; i++) {
	  if (!((row_extracted >> i) & 1L)) {
	    col_sum += d_r[i] * M[i*nov + j];
	  }
	}
	if (col_sum == 0) {
	  return false;
	}
	d_c[j] = 1 / col_sum;
      }
    }
    for (int i = 0; i < nov; i++) {
      if (!((row_extracted >> i) & 1L)) {
	C row_sum = 0;
	for (int j = 0; j < nov; j++) {
	  if (!((col_extracted >> j) & 1L)) {
	    row_sum += M[i*nov + j] * d_c[j];
	  }
	}
	if (row_sum == 0) {
	  return false;
	}
	d_r[i] = 1 / row_sum;
      }
    }
  }
  
  return true;
}
//

template <class C, class S>
  C cpu_rasmussen(S* mat,
		       S* mat_t,
		       int nov,
		       int random,
		       int number_of_times,
		       int threads) {
  
  srand(random);
  
  C sum_perm = 0;
  C sum_zeros = 0;

  
  #pragma omp parallel for num_threads(threads) reduction(+:sum_perm) reduction(+:sum_zeros)
    for (int time = 0; time < number_of_times; time++) {
      int row_nnz[nov];
      long col_extracted = 0;
      
      for (int i = 0; i < nov; i++) {
        row_nnz[i] = 0;
        for (int j = 0; j < nov; j++) {
          if (mat[(i * nov) + j] != (S)0) {
            row_nnz[i] += 1;
          }
        }
      }
      
      C perm = 1;
      
      for (int row = 0; row < nov; row++) {
        // multiply permanent with number of nonzeros in the current row
        perm *= row_nnz[row];

        // choose the column to be extracted randomly
        int random = rand() % row_nnz[row];
        int col;
        for (int c = 0; c < nov; c++) {
          if (!((col_extracted >> c) & 1L) && mat[row * nov + c] != (S)0) {
            if (random == 0) {
              col = c;
              break;
            } else {
              random--;
            }        
          }
        }

        // exract the column
        col_extracted |= (1L << col);

        // update number of nonzeros of the rows after extracting the column
        bool zero_row = false;
        for (int r = row + 1; r < nov; r++) {
          if (mat_t[col * nov + r] != 0) {
            row_nnz[r]--;
            if (row_nnz[r] == 0) {
              zero_row = true;
              break;
            }
          }
        }

        if (zero_row) {
          perm = 0;
          sum_zeros += 1;
          break;
        }
      }

      sum_perm += perm;
    }

    //printf("CPU returning: %f \n", (double)sum_perm);
    return sum_perm;
    
}

template <class C, class S>
  C cpu_approximation_perman64(S* mat,
			       int nov,
			       int random,
			       int number_of_times,
			       int scale_intervals,
			       int scale_times,
			       int threads) {
  
  srand(random);

  C sum_perm = 0;
  C sum_zeros = 0;
    
  #pragma omp parallel for num_threads(threads) reduction(+:sum_perm) reduction(+:sum_zeros)
    for (int time = 0; time < number_of_times; time++) {
      long col_extracted = 0;

      C Xa = 1;
      C d_r[nov];
      C d_c[nov];
      for (int i = 0; i < nov; i++) {
        d_r[i] = 1;
        d_c[i] = 1;
      }
      
      for (int row = 0; row < nov; row++) {
        // Scale part
        if ((scale_intervals != -1 || (scale_intervals == -1 && row == 0)) && row % scale_intervals == 0) {
          bool success = ScaleMatrix(mat, nov, row, col_extracted, d_r, d_c, scale_times);
          if (!success) {
            Xa = 0;
            sum_zeros++;
            break;
          }
        }
        
        // use scaled matrix for pj
        C sum_row_of_S = 0;
        for (int j = 0; j < nov; j++) {
          if (!((col_extracted >> j) & 1L) && mat[(row * nov) + j] != 0) {
            sum_row_of_S += d_r[row] * mat[(row * nov) + j] * d_c[j];
          }
        }
        if (sum_row_of_S == 0) {
          Xa = 0;
          sum_zeros++;
          break;
        }

        C random = (C(rand()) / RAND_MAX) * sum_row_of_S;
        C temp = 0;
        C s, pj;
        int col;
        for (int j = 0; j < nov; j++) {
          if (!((col_extracted >> j) & 1L) && mat[(row * nov) + j] != 0) {
            s = d_r[row] * mat[(row * nov) + j] * d_c[j];
            temp += s;
            if (random <= temp) {
              col = j;
              pj = s / sum_row_of_S;
              break;
            }
          }
        }

        // update Xa
        Xa /= pj;
        
        // exract the column
        col_extracted |= (1L << col);

      }
      
      sum_perm += Xa;
    }

    //xprintf("CPU Returning: %f \n", (double)sum_perm);
    return sum_perm;
}

//GLOBAL MEMORY KERNELS SINCE WE CANNOT KEEP VERY BIG MATRICES IN SHARED MEMORY//
template <class C, class S>
  __global__ void kernel_rasmussen_global(S* mat,
					  C* p,
					  int nov,
					  int rand){
  
  
  int tid = threadIdx.x + (blockIdx.x * blockDim.x);
  int thread_id = threadIdx.x;
  int block_dim = blockDim.x;
  
  hiprandState_t state;
  hiprand_init(rand*tid,0,0,&state);

  long col_extracted = 0;
  long row_extracted = 0;
  
  C perm = 1;
  int row;
  
  for (int i = 0; i < nov; i++) {
    // multiply permanent with number of nonzeros in the current row
    int min_nnz = nov+1;
    int nnz;
    for (int r = 0; r < nov; r++) {
      if (!((row_extracted >> r) & 1L)) {
        nnz = 0;
        for (int c = 0; c < nov; c++) {
          if (!((col_extracted >> c) & 1L) && mat[r * nov + c] != 0) {
            nnz++;
          }
        }
        if (min_nnz > nnz) {
          min_nnz = nnz;
          row = r;
        }
      }
    }
    
    if (min_nnz == 0) {
      perm = 0;
      break;
    }
    perm *= min_nnz;
    
    // choose the column to be extracted randomly
    int random = hiprand_uniform(&state) / (1.0 / ((C)(min_nnz)));
    int col;
    
    if (random >= min_nnz) {
      random = min_nnz - 1;
    }
    for (int c = 0; c < nov; c++) {
      if (!((col_extracted >> c) & 1L) && mat[row * nov + c] != 0) {
        if (random == 0) {
          col = c;
          break;
        } else {
          random--;
        }        
      }
    }
    
    // exract the column
    col_extracted |= (1L << col);
    // exract the row
    row_extracted |= (1L << row);
  }
  
  p[tid] = perm;
  
}

    


int rasmussen_sharedmem(int b){
  //printf("b: %d || glob_nov: %d || glob_sizeof_s: %d \n", b, glob_nov, glob_sizeof_s);
  //printf("--> Will return %d \n", b*glob_nov*glob_nov*glob_sizeof_s);
  return b*glob_nov*glob_nov*glob_sizeof_s;
}

//Actually the same but prevents confusion
int scaling_sharedmem(int b){ 
  return b*glob_nov*glob_nov*glob_sizeof_s;
}

//This unary functions would be the case if we were using a dynamically sized array for every
//gpu kernel but we don't so, we use CudaOccupancyMatPotentialBlockSize

template <class C, class S>
  __global__ void kernel_rasmussen_shared(S* mat,
					  C* p,
					  int nov,
					  int rand) {
  
  int tid = threadIdx.x + (blockIdx.x * blockDim.x);
  int thread_id = threadIdx.x;
  int block_dim = blockDim.x;

  extern __shared__ double shared_mem[]; 
  S *shared_mat = (S*) shared_mem; // size = nov * nov

  for (int k = 0; k < ((nov*nov)/block_dim + 1); k++) {
    if ((block_dim * k + thread_id) < (nov * nov))
      shared_mat[block_dim * k + thread_id] = mat[block_dim * k + thread_id];
  }
  
  __syncthreads();
  
  hiprandState_t state;
  hiprand_init(rand*tid,0,0,&state);
  
  long col_extracted = 0;
  long row_extracted = 0;
  
  C perm = 1;
  int row;
  
  for (int i = 0; i < nov; i++) {
    // multiply permanent with number of nonzeros in the current row
    int min_nnz = nov+1;
    int nnz;
    for (int r = 0; r < nov; r++) {
      if (!((row_extracted >> r) & 1L)) {
        nnz = 0;
        for (int c = 0; c < nov; c++) {
          if (!((col_extracted >> c) & 1L) && shared_mat[r * nov + c] != 0) {
            nnz++;
          }
        }
        if (min_nnz > nnz) {
          min_nnz = nnz;
          row = r;
        }
      }
    }
    
    if (min_nnz == 0) {
      perm = 0;
      break;
    }
    perm *= min_nnz;
    
    // choose the column to be extracted randomly
    int random = hiprand_uniform(&state) / (1.0 / ((C)(min_nnz)));
    int col;
    
    if (random >= min_nnz) {
      random = min_nnz - 1;
    }
    for (int c = 0; c < nov; c++) {
      if (!((col_extracted >> c) & 1L) && shared_mat[row * nov + c] != 0) {
        if (random == 0) {
          col = c;
          break;
        } else {
          random--;
        }        
      }
    }

    // exract the column
    col_extracted |= (1L << col);
    // exract the row
    row_extracted |= (1L << row);
  }

  p[tid] = perm;
}

template <class C, class S>
  __global__ void kernel_approximation(S* mat,
				       C* p,
				       C* d_r,
				       C* d_c,
				       int nov,
				       int scale_intervals,
				       int scale_times,
				       int rand) {
  int tid = threadIdx.x + (blockIdx.x * blockDim.x);
  int thread_id = threadIdx.x;
  int block_dim = blockDim.x;
  
  extern __shared__ double shared_mem[]; 
  S *shared_mat = (S*) shared_mem; // size = nov * nov

  for (int k = 0; k < ((nov*nov)/block_dim + 1); k++) {
    if ((block_dim * k + thread_id) < (nov * nov))
      shared_mat[block_dim * k + thread_id] = mat[block_dim * k + thread_id];
  }
  
  __syncthreads();
  
  hiprandState_t state;
  hiprand_init(rand*tid,0,0,&state);

  long col_extracted = 0;
  long row_extracted = 0;
  bool is_break;
  
  for (int i = 0; i < nov; i++) {
    d_r[tid*nov + i] = 1;
    d_c[tid*nov + i] = 1;
  }
  
  C perm = 1;
  C col_sum, row_sum;
  int row;
  int min;
  int nnz;
  
  for (int iter = 0; iter < nov; iter++) {
    min=nov+1;
    for (int i = 0; i < nov; i++) {
      if (!((row_extracted >> i) & 1L)) {
        nnz = 0;
        for (int j = 0; j < nov; j++) {
          if (!((col_extracted >> j) & 1L) && shared_mat[(i * nov) + j] != 0) {
            nnz++;
          }
        }
        if (min > nnz) {
          min = nnz;
          row = i;
        }
      }
    }
    // Scale part
    if (iter % scale_intervals == 0) {
      
      for (int k = 0; k < scale_times; k++) {
	
        for (int j = 0; j < nov; j++) {
          if (!((col_extracted >> j) & 1L)) {
            col_sum = 0;
            for (int i = 0; i < nov; i++) {
              if (!((row_extracted >> i) & 1L)) {
                col_sum += d_r[tid*nov + i] * shared_mat[i*nov + j];
              }
            }
            if (col_sum == 0) {
              is_break = true;
              break;
            }
            d_c[tid*nov + j] = 1 / col_sum;
          }
        }
        if (is_break) {
          break;
        }
	
        for (int i = 0; i < nov; i++) {
          if (!((row_extracted >> i) & 1L)) {
            row_sum = 0;
            for (int j = 0; j < nov; j++) {
              if (!((col_extracted >> j) & 1L)) {
                row_sum += shared_mat[i*nov + j] * d_c[tid*nov + j];
              }
            }
            if (row_sum == 0) {
              is_break = true;
              break;
            }
            d_r[tid*nov + i] = 1 / row_sum;
          }
        }
        if (is_break) {
          break;
        }
      }
      
    }

    if (is_break) {
      perm = 0;
      break;
    }
    
    // use scaled matrix for pj
    C sum_row_of_S = 0;
    for (int j = 0; j < nov; j++) {
      if (!((col_extracted >> j) & 1L) && shared_mat[(row * nov) + j] != 0) {
        sum_row_of_S += d_r[tid*nov + row] * d_c[tid*nov + j];
      }
    }
    if (sum_row_of_S == 0) {
      perm = 0;
      break;
    }
    
    C random = hiprand_uniform(&state) * sum_row_of_S;
    C temp = 0;
    C s, pj;
    int col;
    for (int j = 0; j < nov; j++) {
      if (!((col_extracted >> j) & 1L) && shared_mat[(row * nov) + j] != 0) {
        s = d_r[tid*nov + row] * d_c[tid*nov + j];
        temp += s;
        if (random <= temp) {
          col = j;
          pj = s / sum_row_of_S;
          break;
        }
      }
    }
    
    // update perm
    perm /= pj;
    
    // exract the column
    col_extracted |= (1L << col);
    // exract the row
    row_extracted |= (1L << row);
  }
  
  p[tid] = perm;
}



template <class C, class S>
  extern Result gpu_perman64_rasmussen(DenseMatrix<S>* densemat, flags flags) {
  
  //Pack parameters//
  S* mat = densemat->mat;
  int nov = densemat->nov;
  //Pack parameters//
  
  //Pack flags//
  int number_of_times = flags.number_of_times;
  int device_id = flags.device_id;
  int grid_dim_multip = flags.grid_multip;
  //Pack flags//
  
  hipSetDevice(device_id);
  hipDeviceSynchronize();
  
  double starttime = omp_get_wtime();
  
  int grid_dim = 1024;
  int block_dim = number_of_times / grid_dim + 1;
  
  glob_nov = nov;
  glob_sizeof_c = sizeof(C);
  glob_sizeof_s = sizeof(S);
  
  size_t size = nov*nov*sizeof(S);
  
  hipOccupancyMaxPotentialBlockSize(&grid_dim,
				     &block_dim,
				     &kernel_rasmussen_shared<C,S>,
				     size,
				     0);

  
  printf("==SC== Shared memory per block is set to : %zu \n", size);
  printf("==SC== Grid dim is set to : %d \n", grid_dim);
  printf("==SC== Block dim is set to : %d \n", block_dim);
  
  if(grid_dim_multip != 1){
    grid_dim*=grid_dim_multip;
    printf("==SC== Grid dim is re-set to : %d \n", grid_dim);
  }

  
  
  S *d_mat;
  C *d_p;
  //Let's use dynamic shared memory and choose grid and block dim according to the matrix size
  //and type
  C *h_p = new C[grid_dim * block_dim];
  
  //hipMalloc( &d_p, (grid_dim * block_dim) * sizeof(C));
  hipMalloc( &d_mat, (nov * nov) * sizeof(S));

  hipMemcpy( d_mat, mat, (nov * nov) * sizeof(S), hipMemcpyHostToDevice);

  srand(time(0));

  double one_run = grid_dim * block_dim;
  double current = 0;

  double p = 0;

  while(current < number_of_times){

    hipMalloc( &d_p, (grid_dim * block_dim) * sizeof(C));
    
    kernel_rasmussen_shared<C,S><<<grid_dim , block_dim , size>>> (d_mat, d_p, nov, rand());
    hipDeviceSynchronize();

    hipMemcpy( h_p, d_p, grid_dim * block_dim * sizeof(C), hipMemcpyDeviceToHost);

    for(int i = 0; i < grid_dim * block_dim; i++){
      p += h_p[i];
    }
    current += one_run;
    hipFree(d_p);
  }

  hipFree(d_mat);
  //hipFree(d_p);
  
  delete[] h_p;

  printf("==SI== Actual Times: %d \n", int(current));
  
  double perman = p / current;
  double duration = omp_get_wtime() - starttime;
  Result result(perman, duration);
  return result;
}

template <class C, class S>
  extern Result gpu_perman64_rasmussen_multigpucpu_chunks(DenseMatrix<S>* densemat, flags flags) {
  
  //Pack parameters//
  S* mat = densemat->mat;
  int nov = densemat->nov;
  //Pack parameters//
  
  //Pack flags//
  int number_of_times = flags.number_of_times;
  int gpu_num = flags.gpu_num;
  bool cpu = flags.cpu;
  int threads = flags.threads;
  int f_grid_dim = flags.grid_dim;
  int f_block_dim = flags.block_dim;
  int grid_dim_multip = flags.grid_multip;
  //Pack flags//
  
  hipDeviceProp_t* props = new hipDeviceProp_t[gpu_num];
  for(int i = 0; i < gpu_num; i++){
    hipGetDeviceProperties(&props[i], i);
    printf("===SC=== Using Device: %d -- %s \n", i, props[i].name);
  }
  
  double starttime = omp_get_wtime();
  int gpu_driver_threads = gpu_num;
  int calculation_threads = threads - gpu_num;
  
  printf("===SC=== Using %d threads for GPU drivers \n", gpu_driver_threads);
  printf("===SC=== Using %d threads for calculation \n", calculation_threads);
  
  if(calculation_threads < 1){
    printf("===WARNING=== No calculation threads left for CPU \n");
    cpu = false;
  }
  
  int grid_dims[gpu_num];
  int block_dims[gpu_num];
  
  //For max potential block size
  glob_nov = nov;
  glob_sizeof_c = sizeof(C);
  glob_sizeof_s = sizeof(S);
  //For max potential block size
  
  size_t size = nov*nov*sizeof(S);
  
  for(int dev = 0; dev < gpu_num; dev++){
    hipSetDevice(dev);
    
    hipOccupancyMaxPotentialBlockSize(&grid_dims[dev],
				       &block_dims[dev],
				       &kernel_rasmussen_shared<C,S>,
				       size,
				       0);

    if(grid_dim_multip != 1){
      grid_dims[dev] *= grid_dim_multip;
    }
  }
  
  int if_cpu = (int)cpu;
  
  unsigned long long cpu_chunk = if_cpu * (number_of_times / 100);
  unsigned long long gpu_chunks[gpu_num];
  
  for(int dev = 0; dev < gpu_num; dev++){
    gpu_chunks[dev] = grid_dims[dev] * block_dims[dev];
  }
  
  C p = 0;
  C p_partial[gpu_num + if_cpu];
  C p_partial_times[gpu_num + if_cpu];
  
  for (int dev = 0; dev < gpu_num + if_cpu; dev++) {
    p_partial[dev] = 0;
    p_partial_times[dev] = 0;
  }
  
  unsigned long long curr_chunk = 0;
  //for(int dev = 0;  dev < gpu_num; dev++){
  //curr_chunk +=  gpu_chunks[dev];
  //}

  //curr_chunk += if_cpu * cpu_chunk;

  curr_chunk = 0; //Note that this is different than exact calculation
  srand(time(0));
  
  omp_set_nested(1);
  omp_set_dynamic(0);
#pragma omp parallel num_threads(gpu_num + if_cpu) 
  {
    
    int tid = omp_get_thread_num();
    int nt = omp_get_num_threads();
    
    unsigned long long last = curr_chunk;
    
#pragma omp barrier
    
    if (tid == gpu_num) {//CPU PART
      
      S* mat_t = new S[nov * nov]; //Create transpose of the matrix
      for (int i = 0; i < nov; i++) {
	for (int j = 0; j < nov; j++) {
	  mat_t[(j * nov) + i] = mat[(i * nov) + j];
	}
      }

      while (last < number_of_times) {
	p_partial[tid] += cpu_rasmussen<C,S>(mat, mat_t, nov, (int)rand(), (int)cpu_chunk, calculation_threads);
	p_partial_times[tid] += cpu_chunk;
	
#pragma omp atomic update
	curr_chunk += cpu_chunk;
#pragma omp atomic read
	last = curr_chunk;
      }	
      delete[] mat_t;
    }//CPU PART
    
    else { //GPU PART
      
      hipSetDevice(tid);
      
      int grid_dim = grid_dims[tid];
      int block_dim = block_dims[tid];
      
      hipStream_t thread_stream;
      hipStreamCreate(&thread_stream);
      
      S *d_mat;
      C *d_p;
      C *h_p = new C[grid_dims[tid] * block_dims[tid]];
      
      //hipMalloc( &d_p, (grid_size * block_size) * sizeof(C));
      hipMalloc( &d_mat, (nov * nov) * sizeof(S));
      
      hipMemcpy( d_mat, mat, (nov * nov) * sizeof(S), hipMemcpyHostToDevice);
      
      while (last < number_of_times) {
	
	hipMalloc(&d_p, (grid_dims[tid] * block_dims[tid]) * sizeof(C));
	kernel_rasmussen_shared<<< grid_dims[tid] , block_dims[tid] , size >>>(d_mat, d_p, nov, rand());
	hipStreamSynchronize(thread_stream);
	hipMemcpy( h_p, d_p, grid_dims[tid] * block_dims[tid] * sizeof(C), hipMemcpyDeviceToHost);
	
	for (int i = 0; i < grid_dims[tid] * block_dims[tid]; i++) {
	  p_partial[tid] += h_p[i];
	}
	
	p_partial_times[tid] += (grid_dims[tid] * block_dims[tid]);
	hipFree(d_p);
	
#pragma omp atomic update
	curr_chunk += gpu_chunks[tid];
#pragma omp atomic read
	last = curr_chunk;
      }
      
      hipFree(d_mat);
      
      delete[] h_p;
    }
  }
  
  for (int dev = 0; dev < gpu_num + if_cpu; dev++) {
    p += p_partial[dev];
  }
  
  double times = 0;
  for (int id = 0; id < gpu_num + if_cpu; id++) {
    times += p_partial_times[id];
  }
  
  double duration = omp_get_wtime() - starttime;
  printf("==SI== Actual Times: %d \n", (int)times);
  
  double perman = p / times;
  Result result(perman, duration);
  return result;
}

template <class C, class S>
extern Result gpu_perman64_approximation(DenseMatrix<S>* densemat, flags flags) {

  //Pack parameters//
  S* mat = densemat->mat;
  int nov = densemat->nov;
  //Pack parameters//

  //Pack flags//
  int scale_intervals = flags.scale_intervals;
  int scale_times = flags.scale_times;
  int number_of_times = flags.number_of_times;
  int device_id = flags.device_id;
  int grid_dim_multip = flags.grid_multip;
  //Pack flags//

  hipSetDevice(device_id);
  hipDeviceSynchronize();

  double starttime = omp_get_wtime();
  
  int block_dim;// = 1024;
  int grid_dim;// = number_of_times / block_size + 1;

  glob_nov = nov;
  glob_sizeof_c = sizeof(C);
  glob_sizeof_s = sizeof(S);

  size_t size = nov*nov*sizeof(S);
  
  hipOccupancyMaxPotentialBlockSize(&grid_dim,
				     &block_dim,
				     &kernel_approximation<C,S>,
				     size,
				     0);
  
  printf("==SC== Shared memory per block is set to: %zu ..\n", size);
  printf("==SC== Grid dim is set to : %d \n", grid_dim);
  printf("==SC== Block dim is set to : %d \n", block_dim);
  
  if(grid_dim_multip != 1){
    grid_dim*=grid_dim_multip;
    printf("==SC== Grid dim is re-set to : %d \n", grid_dim);
  }

  

  C *h_p = new C[grid_dim * block_dim];

  S *d_mat;
  C *d_p;
  C *d_r, *d_c;

  //hipMalloc( &d_p, (grid_dim * block_dim) * sizeof(C));
  hipMalloc( &d_mat, (nov * nov) * sizeof(S));
  
  hipMemcpy( d_mat, mat, (nov * nov) * sizeof(S), hipMemcpyHostToDevice);
  
  srand(time(0));

  hipMalloc( &d_r, (nov * grid_dim * block_dim) * sizeof(C));
  hipMalloc( &d_c, (nov * grid_dim * block_dim) * sizeof(C));


  double one_run = grid_dim * block_dim;
  double current = 0;

  double p = 0;

  while(current < number_of_times){
    hipMalloc( &d_p, (grid_dim * block_dim) * sizeof(C));
    
    kernel_approximation<C,S><<<grid_dim, block_dim, size>>> (d_mat, d_p, d_r, d_c, nov, scale_intervals, scale_times, rand());
    hipDeviceSynchronize();

    hipMemcpy( h_p, d_p, grid_dim * block_dim * sizeof(C), hipMemcpyDeviceToHost);
    
    for (int i = 0; i < grid_dim * block_dim; i++) {
      p += h_p[i];
    }

    current += one_run;
    hipFree(d_p);
  }
  
  
  hipFree(d_mat);
  
  hipFree(d_r);
  hipFree(d_c);

  
  delete[] h_p;

  double perman = p / (grid_dim * block_dim);
  double duration = omp_get_wtime() - starttime;
  Result result(perman, duration);
  return result;

  //return (p / (grid_dim * block_dim));
}

template <class C, class S>
extern Result gpu_perman64_approximation_multigpucpu_chunks(DenseMatrix<S>* densemat, flags flags) {

  //Pack parameters//
  S* mat = densemat->mat;
  int nov = densemat->nov;
  //Pack parameters//

  //Pack flags//
  int number_of_times = flags.number_of_times;
  int gpu_num = flags.gpu_num;
  bool cpu = flags.cpu;
  int scale_intervals = flags.scale_intervals;
  int scale_times = flags.scale_times;
  int threads = flags.threads;
  int f_grid_dim = flags.grid_dim;
  int f_block_dim = flags.block_dim;
  int grid_dim_multip = flags.grid_multip;
  //Pack flags//
  
  hipDeviceProp_t* props = new hipDeviceProp_t[gpu_num];
  for(int i = 0; i < gpu_num; i++){
    hipGetDeviceProperties(&props[i], i);
    printf("==SC== Using Device: %d -- %s \n", i, props[i].name);
  }

  double starttime = omp_get_wtime();
  int gpu_driver_threads = gpu_num;
  int calculation_threads = threads - gpu_num;

  printf("==SC== Using %d threads for GPU drivers \n", gpu_driver_threads);
  printf("==SC== Using %d threads for calculation \n", calculation_threads);

  if(calculation_threads < 1){
    printf("==WARNING== No calculation threads left for CPU \n");
    cpu = false;
  }

  int grid_dims[gpu_num];
  int block_dims[gpu_num];

  //For max potential block size
  glob_nov = nov;
  glob_sizeof_c = sizeof(C);
  glob_sizeof_s = sizeof(S);
  //For max potential block size

  size_t size = nov*nov*sizeof(S);

  for(int dev = 0; dev < gpu_num; dev++){
    hipSetDevice(dev);

    hipOccupancyMaxPotentialBlockSize(&grid_dims[dev],
				       &block_dims[dev],
				       &kernel_approximation<C,S>,
				       size,
				       0);

    if(grid_dim_multip != 1){
      grid_dims[dev] *= grid_dim_multip;
    }
    
  }

  int if_cpu = (int)cpu;

  unsigned long long cpu_chunk = if_cpu * (number_of_times / 100);
  unsigned long long gpu_chunks[gpu_num];

  for(int dev = 0; dev < gpu_num; dev++){
    gpu_chunks[dev] = grid_dims[dev] * block_dims[dev];
  }
  
  C p = 0;
  C p_partial[gpu_num + if_cpu];
  C p_partial_times[gpu_num + if_cpu];
  
  for (int dev = 0; dev < gpu_num + if_cpu; dev++) {
    p_partial[dev] = 0;
    p_partial_times[dev] = 0;
  }

  unsigned long long curr_chunk = 0;
  //for(int dev = 0; dev < gpu_num; dev++){
  //curr_chunk += gpu_chunks[dev];
  //}
  //curr_chunk += if_cpu * cpu_chunk;

  curr_chunk = 0; //Note that this is different than exact calculation
  
  srand(time(0));

  omp_set_nested(1);
  omp_set_dynamic(0);
#pragma omp parallel num_threads(gpu_num + if_cpu)
  {
    
    int tid = omp_get_thread_num();
    int nt = omp_get_num_threads();
    
    unsigned long long last = curr_chunk;
    
#pragma omp barrier
    
    if (tid == gpu_num) {//CPU PART
      
      S* mat_t = new S[nov * nov];
      for (int i = 0; i < nov; i++) {
	for (int j = 0; j < nov; j++) {
	  mat_t[(j * nov) + i] = mat[(i * nov) + j];
	}
      }
      
      while (last < number_of_times) {
	p_partial[tid] += cpu_approximation_perman64<C,S>(mat, nov, rand(), cpu_chunk, scale_intervals, scale_times, calculation_threads);	  
	p_partial_times[tid] += cpu_chunk;
	
#pragma omp atomic update
	curr_chunk += cpu_chunk;
#pragma omp atomic read
	last = curr_chunk;
      }
      
      delete[] mat_t;
      
    } //CPU PART
    else { //GPU PART
      
      hipSetDevice(tid);

      int grid_dim = grid_dims[tid];
      int block_dim = block_dims[tid];

      hipStream_t thread_stream;
      hipStreamCreate(&thread_stream);
      
      C *h_r, *h_c;
      C *h_p = new C[grid_dim * block_dim];
      
      S *d_mat;
      C *d_p;
      C *d_r, *d_c;
      
      //hipMalloc( &d_p, (grid_size * block_size) * sizeof(C));
      hipMalloc( &d_mat, (nov * nov) * sizeof(S));
      hipMemcpy( d_mat, mat, (nov * nov) * sizeof(S), hipMemcpyHostToDevice);
      
      hipMalloc( &d_r, (nov * grid_dim * block_dim) * sizeof(C));
      hipMalloc( &d_c, (nov * grid_dim * block_dim) * sizeof(C));
      
      
      while (last < number_of_times) {
	
	hipMalloc( &d_p, (grid_dim * block_dim) * sizeof(C));
	kernel_approximation<<< grid_dim , block_dim , size >>> (d_mat, d_p, d_r, d_c, nov, scale_intervals, scale_times, rand());
	hipStreamSynchronize(thread_stream);
	
	hipMemcpy( h_p, d_p, grid_dim * block_dim * sizeof(C), hipMemcpyDeviceToHost);
	
	for (int i = 0; i < grid_dim * block_dim; i++) {
	  p_partial[tid] += h_p[i];
	}
	
	p_partial_times[tid] += (grid_dims[tid] * block_dims[tid]);
	hipFree(d_p);

#pragma omp atomic update
	curr_chunk += gpu_chunks[tid];
#pragma omp atomic read
	last = curr_chunk;
      }
      
      hipFree(d_mat);
      hipFree(d_p);
      hipFree(d_r);
      hipFree(d_c);
      
      delete[] h_p;
    }
  }
  
  for (int dev = 0; dev < gpu_num + if_cpu; dev++) {
    p += p_partial[dev];
  }
  
  double times = 0;
  for (int dev = 0; dev < gpu_num + if_cpu; dev++) {
    times += p_partial_times[dev];
  }

  double duration = omp_get_wtime() - starttime;
  printf("==SI== Actual Times: %d \n", (int)times);
  double perman =  p / times;
  Result result(perman, duration);
  return result;
}


//Explicit instantiations required for separate compilation

/////
template extern Result gpu_perman64_rasmussen<float, int>(DenseMatrix<int>* densemat, flags flags);
template extern Result gpu_perman64_rasmussen<double, int>(DenseMatrix<int>* densemat, flags flags);
template extern Result gpu_perman64_rasmussen<float, float>(DenseMatrix<float>* densemat, flags flags);
template extern Result gpu_perman64_rasmussen<double, float>(DenseMatrix<float>* densemat, flags flags);
template extern Result gpu_perman64_rasmussen<float, double>(DenseMatrix<double>* densemat, flags flags);
template extern Result gpu_perman64_rasmussen<double, double>(DenseMatrix<double>* densemat, flags flags);
/////

/////
template extern Result gpu_perman64_rasmussen_multigpucpu_chunks<float, int>(DenseMatrix<int>* densemat, flags flags);
template extern Result gpu_perman64_rasmussen_multigpucpu_chunks<double, int>(DenseMatrix<int>* densemat, flags flags);
template extern Result gpu_perman64_rasmussen_multigpucpu_chunks<float, float>(DenseMatrix<float>* densemat, flags flags);
template extern Result gpu_perman64_rasmussen_multigpucpu_chunks<double, float>(DenseMatrix<float>* densemat, flags flags);
template extern Result gpu_perman64_rasmussen_multigpucpu_chunks<float, double>(DenseMatrix<double>* densemat, flags flags);
template extern Result gpu_perman64_rasmussen_multigpucpu_chunks<double, double>(DenseMatrix<double>* densemat, flags flags);
/////

/////
template extern Result gpu_perman64_approximation<float, int>(DenseMatrix<int>* densemat, flags flags);
template extern Result gpu_perman64_approximation<double, int>(DenseMatrix<int>* densemat, flags flags);
template extern Result gpu_perman64_approximation<float, float>(DenseMatrix<float>* densemat, flags flags);
template extern Result gpu_perman64_approximation<double, float>(DenseMatrix<float>* densemat, flags flags);
template extern Result gpu_perman64_approximation<float, double>(DenseMatrix<double>* densemat, flags flags);
template extern Result gpu_perman64_approximation<double, double>(DenseMatrix<double>* densemat, flags flags);
/////

//Let's wait for Nebula to become available for multi-gpu optimization
/////
template extern Result gpu_perman64_approximation_multigpucpu_chunks<float, int>(DenseMatrix<int>* densemat, flags flags);
template extern Result gpu_perman64_approximation_multigpucpu_chunks<double, int>(DenseMatrix<int>* densemat, flags flags);
template extern Result gpu_perman64_approximation_multigpucpu_chunks<float, float>(DenseMatrix<float>* densemat, flags flags);
template extern Result gpu_perman64_approximation_multigpucpu_chunks<double, float>(DenseMatrix<float>* densemat, flags flags);
template extern Result gpu_perman64_approximation_multigpucpu_chunks<float, double>(DenseMatrix<double>* densemat, flags flags);
template extern Result gpu_perman64_approximation_multigpucpu_chunks<double, double>(DenseMatrix<double>* densemat, flags flags);
/////

//Explicit instantiations required for separate compilation





